#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <cmath>

#include "hip/hip_runtime.h"
#include ""

#include "/home/sine/husky_sim/src/cuda_bayesian/include/cuda_bayesian/defs.h"
#include "/home/sine/husky_sim/src/cuda_bayesian/include/cuda_bayesian/bayesian.cuh"
#include "/home/sine/husky_sim/src/cuda_bayesian/include/cuda_bayesian/utils.h"
#include "/home/sine/husky_sim/src/cuda_bayesian/include/cuda_bayesian/gpu_timer.h"

#include "ros/ros.h"
#include "nav_msgs/OccupancyGrid.h"
#include "sensor_msgs/JointState.h"

#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace std;
using namespace cv;

float velocity = 0.0;

bof::Cell::Cell() {
    for (int i = 0; i < NUM_VELOCITY; ++i) {
        this->xVelocityDistribution[i] = 0;
        this->yVelocityDistribution[i] = 0;
    }
    this->yVelocityDistribution[4] = 1;
    this->occupiedProbability = 0;
    this->xpos = 0;
    this->ypos = 0;
}

__host__ __device__ bof::Cell::Cell(const float xVelocityDistribution[7], const float yVelocityDistribution[7], const float occupiedProbability, const int xpos, const int ypos) {
    for (int i = 0; i < NUM_VELOCITY; ++i) {
        this->xVelocityDistribution[i] = xVelocityDistribution[i];
        this->yVelocityDistribution[i] = yVelocityDistribution[i];
    }

    this->occupiedProbability = occupiedProbability;
    this->xpos = xpos;
    this->ypos = ypos;
}

__host__ __device__ bof::Cell** bof::Cell::getAntecedents(int *antSize, bof::Cell *prevOccGrid, float dt) {
    int stencilHalfWidth = ceil(MAX_VELOCITY * dt);
    *antSize = ((-stencilHalfWidth + ypos > 0 ? stencilHalfWidth : ypos) + (stencilHalfWidth + ypos < GRID_ROWS ? stencilHalfWidth : GRID_ROWS - ypos - 1) + 1) *
        ((-stencilHalfWidth + xpos > 0 ? stencilHalfWidth : xpos) + (stencilHalfWidth + xpos < GRID_COLS ? stencilHalfWidth : GRID_COLS - xpos - 1) + 1);

    Cell **antecedents = new Cell*[*antSize];

    int counter = 0;
    for (int i = -stencilHalfWidth; i <= stencilHalfWidth; ++i) {
        for (int j = -stencilHalfWidth; j <= stencilHalfWidth; ++j) {
            if (i + ypos >= 0 && i + ypos < GRID_ROWS && j + xpos >= 0 && j + xpos < GRID_COLS) {
                antecedents[counter++] = &prevOccGrid[(i + ypos) * GRID_ROWS + (j + xpos)];
            }
        }
    }

    // printf("stencilHalfWidth: %d, dt: %f, Pos: (%d, %d): antSize: %d\n", stencilHalfWidth, dt, xpos, ypos, *antSize);
    return antecedents;
}

__host__ __device__ void bof::Cell::getPrediction(float *alphaO, float *alphaE, const int xVelocity, const int yVelocity, bof::Cell **antecedents, const int antSize, const bof::Cell *prevOccGrid, float dt) {
    *alphaO = *alphaE = 0.0f;

    for (int i = 0; i < antSize; ++i) {
        Cell *cell = antecedents[i];

        *alphaO += (1.0f / antSize) *
            cell->xVelocityDistribution[(xVelocity + MAX_VELOCITY) / VEL_STRIDE] * cell->yVelocityDistribution[(yVelocity + MAX_VELOCITY) / VEL_STRIDE] *
            cell->isReachable(xVelocity, yVelocity, cell, dt) *
            cell->occupiedProbability;

        *alphaE += (1.0f / antSize) *
            cell->xVelocityDistribution[(xVelocity + MAX_VELOCITY) / VEL_STRIDE] * cell->yVelocityDistribution[(yVelocity + MAX_VELOCITY) / VEL_STRIDE] *
            cell->isReachable(xVelocity, yVelocity, cell, dt) *
            (1.0f - cell->occupiedProbability);
    }
}

__host__ __device__ void bof::Cell::getEstimation(float *alphaOccMatrix, float *alphaEmpMatrix, const float lvkSum) {
    for (int i = 0; i < NUM_VELOCITY; ++i) {
        for (int j = 0; j < NUM_VELOCITY; ++j) {
            alphaOccMatrix[i * NUM_VELOCITY + j] /= lvkSum;
            alphaEmpMatrix[i * NUM_VELOCITY + j] /= lvkSum;
        }
    }
}

__host__ __device__ float bof::Cell::getNewOccupiedProbability(const float *alphaOccMatrix) {
    float sum = 0.0f;
    for (int i = 0; i < NUM_VELOCITY; ++i) {
        for (int j = 0; j < NUM_VELOCITY; ++j) {
            sum += alphaOccMatrix[i * NUM_VELOCITY + j];
        }
    }

    return sum;
}

__host__ __device__ void bof::Cell::updateVelocityProbabilities(const float *alphaOccMatrix, const float *alphaEmpMatrix) {
    for (int i = 0; i < NUM_VELOCITY; ++i) {
        for (int j = 0; j < NUM_VELOCITY; ++j) {
            xVelocityDistribution[j] += alphaOccMatrix[i * NUM_VELOCITY + j] + alphaEmpMatrix[i * NUM_VELOCITY + j];
            yVelocityDistribution[i] += alphaOccMatrix[i * NUM_VELOCITY + j] + alphaEmpMatrix[i * NUM_VELOCITY + j];
        }
    }
}

__host__ __device__ int bof::Cell::isReachable(const int xVelocity, const int yVelocity, const bof::Cell *cell, float dt) {
    int reachableXPos = lroundf(cell->xpos + xVelocity * dt);
    int reachableYPos = lroundf(cell->ypos + yVelocity * dt);

    if (xpos == reachableXPos && ypos == reachableYPos)
        return 1;

    return 0;
}

__host__ __device__ void bof::Cell::updateDistributions(bof::Cell *prevOccGrid, float dt) {
    assert(dt > 0);

    int antSize;
    Cell **antecedents = getAntecedents(&antSize, prevOccGrid, dt);
    assert(antSize > 0);

    float *betaOccMatrix = new float[NUM_VELOCITY * NUM_VELOCITY];
    float *betaEmpMatrix = new float[NUM_VELOCITY * NUM_VELOCITY];
    float lvkSum = 0;

    for (int xVel = -MAX_VELOCITY; xVel <= MAX_VELOCITY; xVel += VEL_STRIDE) {
        int i = (xVel + MAX_VELOCITY) / VEL_STRIDE;
        for (int yVel = -MAX_VELOCITY; yVel <= MAX_VELOCITY; yVel += VEL_STRIDE) {
            int j = (yVel + MAX_VELOCITY) / VEL_STRIDE;
            float alphaO = 0, alphaE = 0;
            getPrediction(&alphaO, &alphaE, xVel, yVel, antecedents, antSize, prevOccGrid, dt);

            float betaO = xVelocityDistribution[i] * yVelocityDistribution[j] * alphaO;
            float betaE = xVelocityDistribution[i] * yVelocityDistribution[j] * alphaE;
            lvkSum += betaO + betaE;

            betaOccMatrix[i * NUM_VELOCITY + j] = betaO;
            betaEmpMatrix[i * NUM_VELOCITY + j] = betaE;
        }
    }

    if (!lvkSum) {
        getEstimation(betaOccMatrix, betaEmpMatrix, lvkSum);
        occupiedProbability = getNewOccupiedProbability(betaOccMatrix);
        updateVelocityProbabilities(betaOccMatrix, betaEmpMatrix);
    }

    delete betaOccMatrix;
    delete betaEmpMatrix;
    delete antecedents;
}

void bof::Cell::toString() {
    std::cout << "Pos: (" << xpos << ", " << ypos << "), ";

    std::cout << "xVel: [" << xVelocityDistribution[0];
    for (int i = 1; i < NUM_VELOCITY; ++i)
        std::cout << ", " << xVelocityDistribution[i];

    std::cout << "], yVel: [" << yVelocityDistribution[0];
    for (int i = 1; i < NUM_VELOCITY; ++i)
        std::cout << ", " << yVelocityDistribution[i];

    std::cout << "], Occ: " << occupiedProbability << std::endl;
}

__global__ void computeDistributions(bof::Cell *occGrid, bof::Cell *prevOccGrid, float dt) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= GRID_COLS || y >= GRID_ROWS) {
        return;
    }

    unsigned int index = y * GRID_ROWS + x;
    occGrid[index].updateDistributions(prevOccGrid, dt);
}

void callKernel(bof::Cell *occGrid, bof::Cell *prevOccGrid, float dt) {
    /* Compute gridsize and blocksize */
    const unsigned int width = 32;
    const dim3 blockSize(width, width, 1);

    unsigned int gridCols = GRID_COLS / width + (GRID_COLS % width != 0);
    unsigned int gridRows = GRID_ROWS / width + (GRID_ROWS % width != 0);
    const dim3 gridSize(gridRows, gridCols, 1);

    computeDistributions<<<gridSize, blockSize>>>(occGrid, prevOccGrid, dt);
}

void printOccupancy(bof::Cell *m) {
    for (int i = 0; i < GRID_ROWS; ++i) {
        for (int j = 0; j < GRID_COLS; ++j) {
            m[i * GRID_ROWS + j].toString();
        }
    }
}


void chatterCallback(const nav_msgs::OccupancyGrid::ConstPtr& msg){
    size_t numBytes = sizeof(bof::Cell) * GRID_ROWS * GRID_COLS;

    bof::Cell *h_m = new bof::Cell[GRID_ROWS * GRID_COLS];
    bof::Cell *d_prevOccGrid;
    bof::Cell *d_occGrid;

    cout << "Size of grid (bytes): " << numBytes << endl;

    for (int i = 0; i < GRID_ROWS; ++i) {
        for (int j = 0; j < GRID_COLS; ++j) {
            h_m[i * GRID_ROWS + j].xpos = j;
            h_m[i * GRID_ROWS + j].ypos = i;
            h_m[i * GRID_ROWS + j].xVelocityDistribution[NUM_VELOCITY / 2] = 1;
            h_m[i * GRID_ROWS + j].yVelocityDistribution[NUM_VELOCITY / 2] = 1;
            h_m[i * GRID_ROWS + j].occupiedProbability = msg->data[i*GRID_ROWS+j];
        }
    }
    // printOccupancy(h_m);
    
    Mat map_without_bayesian = Mat::zeros(GRID_ROWS, GRID_COLS, CV_8UC3);
     for (int i = 0; i < GRID_ROWS; ++i)
	{
	    for (int j = 0; j < GRID_COLS; ++j)
		{
			int a = h_m[i * GRID_ROWS + j].occupiedProbability;
			    if(a == 0 || a == 1)
			    {
				map_without_bayesian.at<Vec3b>(i, j)[0] = a * 255;
			 	map_without_bayesian.at<Vec3b>(i, j)[1] = a * 255;
			 	map_without_bayesian.at<Vec3b>(i, j)[2] = a * 255;
			 }
			}
	}

    checkCudaErrors((hipMalloc(&d_prevOccGrid, numBytes)));
    checkCudaErrors(hipMemcpy(d_prevOccGrid, h_m, numBytes, hipMemcpyHostToDevice));
    checkCudaErrors((hipMalloc(&d_occGrid, numBytes)));
    checkCudaErrors(hipMemcpy(d_occGrid, h_m, numBytes, hipMemcpyHostToDevice));

    /* Call kernel */
    GpuTimer timer;
    timer.Start();
    float dt = 0.5f; // compute time delay here
    callKernel(d_occGrid, d_prevOccGrid, dt);
    timer.Stop();
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    cout << "Elapsed time: " << timer.Elapsed() << endl;
    checkCudaErrors(hipMemcpy(h_m, d_occGrid, numBytes, hipMemcpyDeviceToHost));

    Mat map_with_bayesian = Mat::zeros(GRID_ROWS, GRID_COLS, CV_8UC3);
     for (int i = 0; i < GRID_ROWS; ++i)
	{
	    for (int j = 0; j < GRID_COLS; ++j)
		{
			int a = h_m[i * GRID_ROWS + j].occupiedProbability;
			    if(a == 0 || a == 1)
			    {
				map_with_bayesian.at<Vec3b>(i, j)[0] = a * 255;
			 	map_with_bayesian.at<Vec3b>(i, j)[1] = a * 255;
			 	map_with_bayesian.at<Vec3b>(i, j)[2] = a * 255;
			 }
			}
	}

    waitKey(1);
    imshow("map_with_bayesian", map_with_bayesian);
    imshow("map_without_bayesian", map_without_bayesian);

    // printOccupancy(h_m);
    /* cleanup */
    hipFree(d_prevOccGrid);
    hipFree(d_occGrid);
    delete h_m;

}


int main(int argc, char **argv)
{
  ros::init(argc, argv, "cuda_map");

  ros::NodeHandle n;
  ros::NodeHandle nh;
  ros::NodeHandle nh1;
  ros::Rate loop_rate(25); 

  namedWindow("map_without_bayesian", CV_WINDOW_NORMAL);
  namedWindow("map_with_bayesian", CV_WINDOW_NORMAL);
  
  ros::Subscriber sub = n.subscribe("/scan/fusedOccGrd", 1, chatterCallback);

  while(ros::ok())
    {
        ros::spinOnce();
        loop_rate.sleep();    
    }
  return 0;
}

